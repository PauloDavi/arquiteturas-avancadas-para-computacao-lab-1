#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime.h"
#include ""

using namespace std;

__device__ double d_pi;

double f(double x)
{
  return 4.0 / (1.0 + x * x);
}

__global__ void add_vectors(double *a, double size) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < size) a[id] = f(a[id]);

	typedef hipcub::BlockReduce<double, size> BlockReduce;

	__shared__ typename BlockReduce::TempStorage temp_storage;

	d_pi = BlockReduce(temp_storage).Sum(a);	
}

int main(int argc, char *argv[])
{
  if (argc != 2)
  {
    cout << "Usage: " << argv[0] << " <delta x>" << endl;
    return 1;
  }

  double array_size = atof(argv[1]);
  dx = 1.0 / array_size;

  size_t bytes = array_size * sizeof(double);

  double *A = (double *)malloc(bytes);

  double *d_A;
  hipMalloc(&d_A, bytes);

  for (int i = 0; i < array_size; i++) {
    A[i] = i * dx;
  }

  hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
  int thr_per_blk = 256;
  int blk_in_grid = ceil(float(N) / thr_per_blk);

	add_vectors<<<blk_in_grid, thr_per_blk>>>(d_A, array_size);

	free(A);
  hipFree(d_A);

	typeof(d_pi) pi;
  hipMemcpyFromSymbol(&pi, HIP_SYMBOL("d_pi"), sizeof(pi), 0, hipMemcpyDeviceToHost);
	pi *= dx;
	cout << "Valor da integral de pi: " << setprecision(10) << pi << endl;

  return 0;
}