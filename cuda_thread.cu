// Using CUDA device to calculate pi

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <iomanip>

#define NUM_BLOCK 512	 // Number of thread blocks
#define NUM_THREAD 512 // Number of threads per block

using namespace std;

// Kernel that executes on the CUDA device
__global__ void cal_pi(double *sum, int n, double step)
{
	// Sequential thread index across the blocks
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = idx; i < n; i += NUM_THREAD * NUM_BLOCK)
	{
		double x = (i + 0.5) * step;
		sum[idx] += 4.0 / (1.0 + x * x);
	}
}

// Main routine that executes on the host
int main(int argc, char *argv[])
{
	if (argc != 2)
	{
		cout << "Usage: " << argv[0] << " <delta x>" << endl;
		return 1;
	}

	double n = atoi(argv[1]);
	double step = 1 / n;
	size_t size = NUM_BLOCK * NUM_THREAD * sizeof(double); // Array memory size
	double *sumDev;
	// Allocate array on device
	hipMalloc((void **)&sumDev, size);
	// Initialize array in device to 0
	hipMemset(sumDev, 0, size);
	// Do calculation on device
	cal_pi<<<NUM_BLOCK, NUM_THREAD>>>(sumDev, n, step); // call CUDA kernel

	// Retrieve result from device and store it in host array
	double *sumHost = (double *)malloc(size);
	hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);

	double pi = 0;
	for (int tid = 0; tid < NUM_THREAD * NUM_BLOCK; tid++)
		pi += sumHost[tid];
	pi *= step;

	// Print results
	cout << "Valor da integral de pi: " << setprecision(15) << pi << endl;

	// Cleanup
	free(sumHost);
	hipFree(sumDev);

	return 0;
}